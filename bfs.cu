#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
BFS using Bloom Tree. 
Input format: ./bfs filename num_vertices num_bits num_hash_functions 
--------------------------------------------------------------------*/

#include <bits/stdc++.h>
#include "bloomtree.cu"
using namespace std;

#define l int
#define DISPLAY_BFS_DIST true
#define N_THREADS_PER_BLOCK 1024
const l N = 2e5 + 5;
const l INF = 1e8;



queue<int> q;
// Initialises bfs_dist to INF
__global__ void Init(int bfs_dist[N]) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	bfs_dist[i] = INF;
}

/*__global__ void parallelize_level(queue<int> q,int adj[N],int u,int bfs_dist[N])
{
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if(bfs_dist[adj[tid]] == INT_MAX)
	{
		bfs_dist[adj[tid]] = bfs_dist[u] + 1;
		q.push(adj[tid]);
	}
}*/

/*__global__ void print(int bfs_dist[N])
{
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (bfs_dist[tid] != INF)
	{
		cout << tid << " - " << bfs_dist[tid] << "\n";
	}
}*/

void BreadthFirstSearch(int s) {
	l bfs_dist[num_vertices],*device_bfs_dist;
	hipMalloc(&device_bfs_dist, num_vertices * sizeof(int));
	int n_blocks = (num_vertices+N_THREADS_PER_BLOCK-1)/N_THREADS_PER_BLOCK;
	Init<<<n_blocks,N_THREADS_PER_BLOCK>>>(device_bfs_dist);
	hipMemcpy(bfs_dist, device_bfs_dist, sizeof(int) * num_vertices, hipMemcpyDeviceToHost);
	q.push(s);
	bfs_dist[s] = 0;
	bool *neigh, *adj;
	hipMalloc(&neigh, num_vertices * sizeof(bool));
	adj = (bool *)malloc(num_vertices * sizeof(bool));
	
	while(!q.empty()){
		cout<<"HI";
		int u = q.front();
		q.pop();
		
		InitAllToFalse<<<(num_vertices/1024 + 1), 1024>>>(neigh, num_vertices);
		GetNeighbours(u, neigh);
		hipMemcpy(adj, neigh, num_vertices * sizeof(bool), hipMemcpyDeviceToHost);
		
		vector<int> vert_adj;
		for(int i=0;i<num_vertices;i++)
		{
			if(adj[i]==true)
			{
				cout<<u<<" "<<i<<"\n";
				vert_adj.push_back(i);
			}
		}
		cout<<"\n";
		for (l i = 0; i < vert_adj.size(); ++i) {
			if (bfs_dist[vert_adj[i]] == INF){
				bfs_dist[vert_adj[i]] = bfs_dist[u] + 1;
				q.push(vert_adj[i]);
			}
		}
	}

	if (DISPLAY_BFS_DIST) {
		for (int i = 0; i < num_vertices; ++i) {
			if (bfs_dist[i] != INF) {
				cout << i << " - " << bfs_dist[i] << "\n";
			}
		}
	}

}

int main(int argc,char** argv){
	if (argc != 5) {
		cout << "Input format: ./program_name filename num_vertices num_bits num_hash_functions\n";
		return 0;
	}
	
	clock_t ti;
	
	num_vertices = atoi(argv[2]);
	int num_bits = atoi(argv[3]);
	int num_hash_funs = atoi(argv[4]);
	
	InitBloomTree(num_vertices,num_bits,num_hash_funs);

	ifstream fin(argv[1], ios::in);
	while(!fin.eof()){
		l u,v;
		fin >> u >> v;
		if (u == v) continue;
		AddEdge(u,v);
	}
	fin.close();

	ti = clock();
	BreadthFirstSearch(0);
	ti = clock() - ti;
	
	printf("%.5f\n", float(ti) / CLOCKS_PER_SEC);
	return 0;
}
