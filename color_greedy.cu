#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "bloomtree.cu"
using namespace std;

#define l int
#define DISPLAY_COLORS true
const l N = INT_MAX;
#define N_THREADS_PER_BLOCK 1024

int adj_size;

__global__ void assign(bool is_neighbour_colour[N],int color[N])
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	is_neighbour_colour[tid] = false; 
	color[tid]=-1;
}

__global__ void check(bool is_neighbour_color[N],int color[N],int adj[N])
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;;
	if (color[adj[i]] != -1)
		is_neighbour_color[color[adj[i]]] = true;
}

void Colouring(){
	int *color = new l[num_vertices],*dcolor;
	bool *is_neighbour_color = new bool[num_vertices],*dneigh;
	
	// Initially, there is no color to any neighbour and to any vertex
	int n_blocks = (num_vertices+N_THREADS_PER_BLOCK-1)/N_THREADS_PER_BLOCK;
	hipMemcpy(dcolor, color, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dneigh, is_neighbour_color, sizeof(int) * N, hipMemcpyHostToDevice);
	assign<<<n_blocks,N_THREADS_PER_BLOCK>>>(is_neighbour_color,color);
	hipMemcpy(color, dcolor, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(is_neighbour_color, dneigh, sizeof(int) * N, hipMemcpyDeviceToHost);
	color[0] = 1;
	int num_colors_used = 1;

	for (int v = 1; v < num_vertices; ++v)
	{
		bool t_adj[N];
		GetNeighbours(v, t_adj);
		int adj[N],j=0,*dadj;
		for(int i=0;i<num_vertices;i++)
		{
			if(t_adj[i]==true)
			{
				adj[j]=i;
				j++;
			}
		}
		int block = (j+N_THREADS_PER_BLOCK-1)/N_THREADS_PER_BLOCK;
		int adj_size = j;
		hipMemcpy(dcolor, color, sizeof(int) * N, hipMemcpyHostToDevice);
		hipMemcpy(dneigh, is_neighbour_color, sizeof(int) * N, hipMemcpyHostToDevice);
		hipMemcpy(dadj, adj, sizeof(int) * N, hipMemcpyHostToDevice);
		check<<<block,N_THREADS_PER_BLOCK>>>(is_neighbour_color,color,adj);
		hipMemcpy(color, dcolor, sizeof(int) * N, hipMemcpyDeviceToHost);
		hipMemcpy(adj, dadj, sizeof(int) * N, hipMemcpyDeviceToHost);
		/*for (l i = 0; i < adj.size(); ++i) {
			if (color[adj[i]] != -1)
				is_neighbour_color[color[adj[i]]] = true;
		}*/

		// Finding first unassigned colour
		l c;
		for (c = 1; c <= num_vertices; ++c) {
			if (is_neighbour_color[c] == false) break;
		}
		color[v] = c;

		for (l i = 0; i < j; ++i) {
			if (color[adj[i]] != -1)
				is_neighbour_color[color[adj[i]]] = false;
		}

		if(c > num_colors_used) num_colors_used = c;
	}
	
	if (DISPLAY_COLORS) {
		for(l v = 0; v < num_vertices; ++v) {
			cout << v << " - " << color[v] << "\n";
		}
	}

	cout << "Number of colours used: " << num_colors_used << "\n";
}
int main(int argc, char** argv){
	if (argc != 5) {
		cout << "Input format: ./program_name filename num_vertices num_bits num_hash_functions\n";
		return 0;
	}
	
	clock_t ti;
	
	num_vertices = atoi(argv[2]);
	int num_bits = atoi(argv[3]);
	int num_hash_funs = atoi(argv[4]);
	
	InitBloomTree(num_vertices,num_bits,num_hash_funs);

	ifstream fin(argv[1], ios::in);
	while(!fin.eof()){
		l u,v;
		fin >> u >> v;
		if (u == v) continue;
		AddEdge(u,v);
	}
	fin.close();

	ti = clock();
	Colouring();
	ti = clock() - ti;
	
	printf("%.5f\n", float(ti) / CLOCKS_PER_SEC);
	return 0;
}